#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

void printArray_cuda(int n, int *a, bool abridged = false)
{
    printf("    [ ");
    for (int i = 0; i < n; i++)
    {
        if (abridged && i + 2 == 15 && n > 16)
        {
            i = n - 2;
            printf("... ");
        }
        printf("%3d ", a[i]);
    }
    printf("]\n");
}

namespace StreamCompaction
{
    namespace Efficient
    {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer &timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void up_sweep(const int n, int d, int *idata)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int stride = 1 << (d + 1);
            if (index >= n || index % stride != 0)
            {
                return;
            }
            idata[index + stride - 1] += idata[index + (1 << d) - 1];
        }

        __global__ void down_sweep(const int n, int d, int *idata)
        {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int stride = 1 << (d + 1);
            if (index >= n || index % stride != 0)
            {
                return;
            }
            int left = 1 << d;
            int t = idata[index + left - 1];
            idata[index + left - 1] = idata[index + stride - 1];
            idata[index + stride - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata)
        {
            const int blockSize = 128;
            dim3 blocksPerData((n + blockSize - 1) / blockSize);

            const int arrLen = n * sizeof(int);
            const int paddedN = 1 << ilog2ceil(n);
            const int paddedLen = paddedN * sizeof(int);

            int *devData;
            hipMalloc((void **)&devData, paddedLen);
            hipMemcpy(devData, idata, arrLen, hipMemcpyHostToDevice);
            hipMemset(devData + n, 0, paddedLen - arrLen);

            // timer().startGpuTimer();
            // TODO
            for (int d = 0; d <= log2(paddedN) - 1; d++)
            {
                up_sweep<<<blocksPerData, blockSize>>>(paddedN, d, devData);
            }

            // x[n-1] = 0
            {
                int zero = 0;
                hipMemcpy(devData + paddedN - 1, &zero, sizeof(int), hipMemcpyHostToDevice);
            }

            for (int d = log2(paddedN) - 1; d >= 0; d--)
            {
                down_sweep<<<blocksPerData, blockSize>>>(paddedN, d, devData);
            }
            // timer().endGpuTimer();

            hipMemcpy(odata, devData, arrLen, hipMemcpyDeviceToHost);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata)
        {
            const int blockSize = 128;
            dim3 blocksPerData((n + blockSize - 1) / blockSize);

            const int arrSize = n * sizeof(int);

            int *devInData, *devOutData,*bools, *indices;
            hipMalloc((void **)&devInData, arrSize);
            hipMalloc((void **)&devOutData, arrSize);
            hipMalloc((void **)&bools, arrSize);
            hipMalloc((void **)&indices, arrSize);

            hipMemcpy(devInData, idata, arrSize, hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            Common::kernMapToBoolean<<<blocksPerData, blockSize>>>(n, bools, devInData);
            scan(n, indices, bools);
            Common::kernScatter<<<blocksPerData, blockSize>>>(n, devOutData, devInData, bools, indices);

            timer().endGpuTimer();

            hipMemcpy(odata, devOutData, arrSize, hipMemcpyDeviceToHost);

            int lastBool;
            int lastIndex;
            hipMemcpy(&lastBool, bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastIndex, indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);

            hipFree(bools);
            hipFree(indices);

            return lastIndex + lastBool;
        }
    }
}
