#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax(a, b) (((a) > (b)) ? (a) : (b))
#endif

#ifndef imin
#define imin(a, b) (((a) < (b)) ? (a) : (b))
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

#define NEIGHBOR_SEARCH_SIZE_8 0

/**
 * Check for CUDA errors; print and exit if there was a problem.
 */
void checkCUDAError(const char *msg, int line = -1)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		if (line >= 0)
		{
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

/*****************
 * Configuration *
 *****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
 * Kernel state (pointers are device pointers) *
 ***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices;  // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_sorted_pos;
glm::vec3 *dev_sorted_vel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
 * initSimulation *
 ******************/

__host__ __device__ unsigned int hash(unsigned int a)
{
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

/**
 * LOOK-1.2 - this is a typical helper function for a CUDA kernel.
 * Function for generating a random vec3.
 */
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index)
{
	thrust::default_random_engine rng(hash((int)(index * time)));
	thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

	return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
 * LOOK-1.2 - This is a basic CUDA kernel.
 * CUDA kernel for generating boids with a specified mass randomly around the star.
 */
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 *arr, float scale)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N)
	{
		glm::vec3 rand = generateRandomVec3(time, index);
		arr[index].x = scale * rand.x;
		arr[index].y = scale * rand.y;
		arr[index].z = scale * rand.z;
	}
}

/**
 * Initialize memory, update some globals
 */
void Boids::initSimulation(int N)
{
	numObjects = N;
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	// LOOK-1.2 - This is basic CUDA memory management and error checking.
	// Don't forget to hipFree in  Boids::endSimulation.
	hipMalloc((void **)&dev_pos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void **)&dev_vel1, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

	hipMalloc((void **)&dev_vel2, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

	// LOOK-1.2 - This is a typical CUDA kernel invocation.
	kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
																 dev_pos, scene_scale);
	checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

	// LOOK-2.1 computing grid params
#if NEIGHBOR_SEARCH_SIZE_8
	gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
#else
	gridCellWidth = std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
#endif
	int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
	gridSideCount = 2 * halfSideCount;

	gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	gridInverseCellWidth = 1.0f / gridCellWidth;
	float halfGridWidth = gridCellWidth * halfSideCount;
	gridMinimum.x -= halfGridWidth;
	gridMinimum.y -= halfGridWidth;
	gridMinimum.z -= halfGridWidth;

	// TODO-2.1 TODO-2.3 - Allocate additional buffers here.

	hipMalloc((void **)&dev_particleArrayIndices, N * sizeof(int));
	hipMalloc((void **)&dev_particleGridIndices, N * sizeof(int));

	hipMalloc((void **)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
	hipMalloc((void **)&dev_gridCellEndIndices, gridCellCount * sizeof(int));

	dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
	dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);

	hipMalloc((void **)&dev_sorted_pos, N * sizeof(glm::vec3));
	hipMalloc((void **)&dev_sorted_vel, N * sizeof(glm::vec3));

	hipDeviceSynchronize();
}

/******************
 * copyBoidsToVBO *
 ******************/

/**
 * Copy the boid positions into the VBO so that they can be drawn by OpenGL.
 */
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N)
	{
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N)
	{
		vbo[4 * index + 0] = vel[index].x + 0.3f;
		vbo[4 * index + 1] = vel[index].y + 0.3f;
		vbo[4 * index + 2] = vel[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

/**
 * Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
 */
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities)
{
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

	checkCUDAErrorWithLine("copyBoidsToVBO failed!");

	hipDeviceSynchronize();
}

/******************
 * stepSimulation *
 ******************/

/**
 * LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
 * __device__ code can be called from a __global__ context
 * Compute the new velocity on the body with index `iSelf` due to the `N` boids
 * in the `pos` and `vel` arrays.
 */
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel)
{
	const glm::vec3 self_pos = pos[iSelf];
	glm::vec3 rule1_v;
	glm::vec3 rule2_v;
	glm::vec3 rule3_v;

	glm::vec3 perceived_center = glm::vec3(0.0f, 0.0f, 0.0f);
	int rule_1_count = 0;

	glm::vec3 c = glm::vec3(0.0f, 0.0f, 0.0f);

	glm::vec3 perceived_velocity = glm::vec3(0.0f, 0.0f, 0.0f);
	int rule_3_count = 0;

	for (int i = 0; i < N; i++)
	{
		if (i == iSelf)
		{
			continue;
		}

		float distance = glm::distance(self_pos, pos[i]);

		// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
		if (distance < rule1Distance)
		{
			perceived_center += pos[i];
			rule_1_count++;
		}

		// Rule 2: boids try to stay a distance d away from each other
		if (distance < rule2Distance)
		{
			c -= pos[i] - self_pos;
		}

		// Rule 3: boids try to match the speed of surrounding boids
		if (distance < rule3Distance)
		{
			perceived_velocity += vel[i];
			rule_3_count++;
		}
	}

	if (rule_1_count != 0)
	{
		perceived_center /= rule_1_count;
		rule1_v = (perceived_center - self_pos) * rule1Scale;
	}

	rule2_v = c * rule2Scale;

	if (rule_3_count != 0)
	{
		perceived_velocity /= rule_3_count;
		rule3_v = perceived_velocity * rule3Scale;
	}

	return vel[iSelf] + rule1_v + rule2_v + rule3_v;
}

/**
 * TODO-1.2 implement basic flocking
 * For each of the `N` bodies, update its position based on its current velocity.
 */
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
											 glm::vec3 *vel1, glm::vec3 *vel2)
{
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}

	// Compute a new velocity based on pos and vel1
	glm::vec3 new_vel = computeVelocityChange(N, index, pos, vel1);

	// Clamp the speed
	if (glm::length(new_vel) > maxSpeed)
	{
		new_vel = new_vel / glm::length(new_vel) * maxSpeed;
	}

	// Record the new velocity into vel2. Question: why NOT vel1?
	vel2[index] = new_vel;
}

/**
 * LOOK-1.2 Since this is pretty trivial, we implemented it for you.
 * For each of the `N` bodies, update its position based on its current velocity.
 */
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel)
{
	// Update position by velocity
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N)
	{
		return;
	}
	glm::vec3 thisPos = pos[index];
	thisPos += vel[index] * dt;

	// Wrap the boids around so we don't lose them
	thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
	thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
	thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

	thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
	thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
	thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

	pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution)
{
	return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
								   glm::vec3 gridMin, float inverseCellWidth,
								   glm::vec3 *pos, int *indices, int *gridIndices)
{
	// TODO-2.1
	// - Label each boid with the index of its grid cell.
	// - Set up a parallel array of integer indices as pointers to the actual
	//   boid data in pos and vel1/vel2
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
	{
		return;
	}
	glm::vec3 posInGridSpace = glm::floor((pos[index] - gridMin) * inverseCellWidth);
	int gridIndex = gridIndex3Dto1D(posInGridSpace.x, posInGridSpace.y, posInGridSpace.z, gridResolution);
	gridIndices[index] = gridIndex;
	indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N)
	{
		intBuffer[index] = value;
	}
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
										 int *gridCellStartIndices, int *gridCellEndIndices)
{
	// TODO-2.1
	// Identify the start point of each cell in the gridIndices array.
	// This is basically a parallel unrolling of a loop that goes
	// "this index doesn't match the one before it, must be a new cell!"
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
	{
		return;
	}

	int gridIndex = particleGridIndices[index];

	if (index == 0)
	{
		gridCellStartIndices[gridIndex] = index;
	}
	else if (index == N - 1)
	{
		gridCellEndIndices[gridIndex] = index;
	}
	else if (gridIndex != particleGridIndices[index - 1])
	{
		gridCellStartIndices[gridIndex] = index;
		gridCellEndIndices[particleGridIndices[index - 1]] = index - 1;
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	int *particleArrayIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2)
{
	// TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
	// the number of boids that need to be checked.
	// - Identify the grid cell that this particle is in
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
	{
		return;
	}
	index = particleArrayIndices[index];

	glm::vec3 posInGridSpace = (pos[index] - gridMin) * inverseCellWidth;
	glm::ivec3 intPosInGridSpace = glm::floor(posInGridSpace);
	glm::vec3 fractPosInGridSpace = glm::fract(posInGridSpace);

	// - Identify which cells may contain neighbors. This isn't always 8.
	int minCells[3] = {}; // minX/Y/Z
	int maxCells[3] = {}; // maxX/Y/Z

	for (int i = 0; i < 3; ++i)
	{
		int start, end;
#if NEIGHBOR_SEARCH_SIZE_8
		int gridOffset = 1;
		if (fractPosInGridSpace[i] < 0.5f)
		{
			gridOffset = -1;
		}
		start = intPosInGridSpace[i];
		end = intPosInGridSpace[i] + gridOffset;
#else
		start = intPosInGridSpace[i] - 1;
		end = intPosInGridSpace[i] + 1;
#endif

		// clamp cell search bounds
		minCells[i] = imax(0, imin(start, end));
		maxCells[i] = imin(gridResolution - 1, imax(start, end));
	}

	// Initialize new velocity components based on boid rules
	glm::vec3 cohesionCenter(0.0f);
	int numCohesionNeighbors = 0;

	glm::vec3 separationCenter(0.0f);

	glm::vec3 alignmentVel(0.0f);
	int numAlignmentNeighbors = 0;

	int lastGridIdx = (gridResolution * gridResolution * gridResolution) - 1;
	for (int z = minCells[2]; z <= maxCells[2]; ++z)
	{
		for (int y = minCells[1]; y <= maxCells[1]; ++y)
		{
			for (int x = minCells[0]; x <= maxCells[0]; ++x)
			{
				// - For each cell, read the start/end indices in the boid pointer array.
				int gridIdx = gridIndex3Dto1D(x, y, z, gridResolution);
				if (gridIdx < 0 || gridIdx > lastGridIdx)
				{
					continue; // skip if past cell bounds
				}

				int start = gridCellStartIndices[gridIdx];
				int end = gridCellEndIndices[gridIdx];
				if (start == -1 || end == -1)
				{
					continue;
				}

				// - Access each boid in the cell and compute velocity change from
				//   the boids rules, if this boid is within the neighborhood distance.
				for (int i = start; i <= end; ++i)
				{
					int neighborIdx = particleArrayIndices[i];
					if (neighborIdx == index)
					{
						continue;
					}

					glm::vec3 neighborPos = pos[neighborIdx];
					float distance = glm::distance(pos[index], neighborPos);
					// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
					if (distance < rule1Distance)
					{
						cohesionCenter += neighborPos;
						numCohesionNeighbors++;
					}

					// Rule 2: boids try to stay a distance d away from each other, including other boids
					if (distance < rule2Distance)
					{
						separationCenter -= (neighborPos - pos[index]);
					}

					// Rule 3: boids try to match the speed of surrounding boids
					if (distance < rule3Distance)
					{
						alignmentVel += vel1[neighborIdx];
						numAlignmentNeighbors++;
					}
				}
			}
		}
	}

	glm::vec3 new_vel = vel1[index];

	// Rule 1: Cohesion
	if (numCohesionNeighbors > 0)
	{
		cohesionCenter /= numCohesionNeighbors;
		new_vel += (cohesionCenter - pos[index]) * rule1Scale;
	}

	// Rule 2: Separation
	new_vel += separationCenter * rule2Scale;

	// Rule 3: Alignment
	if (numAlignmentNeighbors > 0)
	{
		alignmentVel /= numAlignmentNeighbors;
		new_vel += alignmentVel * rule3Scale;
	}

	// - Clamp the speed change before putting the new speed in vel2
	if (glm::length(new_vel) > maxSpeed)
	{
		new_vel = new_vel / glm::length(new_vel) * maxSpeed;
	}

	vel2[index] = new_vel;
}

// Kernel to rearrange boid data based on sorted indices
__global__ void kernRearrangeBoidData(
	int N, int *particleArrayIndices, glm::vec3 *pos, glm::vec3 *vel,
	glm::vec3 *pos_sorted, glm::vec3 *vel_sorted)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
	{
		return;
	}

	int sortedIndex = particleArrayIndices[index];

	pos_sorted[index] = pos[sortedIndex];
	vel_sorted[index] = vel[sortedIndex];
}

__global__ void kernUpdateVelNeighborSearchCoherent(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int *gridCellStartIndices, int *gridCellEndIndices,
	glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2)
{
	// TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
	// except with one less level of indirection.
	// This should expect gridCellStartIndices and gridCellEndIndices to refer
	// directly to pos and vel1.

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N)
	{
		return;
	}

	glm::vec3 selfPos = pos[index];
	glm::vec3 posInGridSpace = (selfPos - gridMin) * inverseCellWidth;
	glm::ivec3 intPosInGridSpace = glm::floor(posInGridSpace);
	glm::vec3 fractPosInGridSpace = glm::fract(posInGridSpace);

	glm::vec3 cohesionCenter(0.0f);
	int numCohesionNeighbors = 0;

	glm::vec3 separationCenter(0.0f);

	glm::vec3 alignmentVel(0.0f);
	int numAlignmentNeighbors = 0;

	int lastGridIdx = (gridResolution * gridResolution * gridResolution) - 1;

	float maxDistance = imax(imax(rule1Distance, rule2Distance), rule3Distance);

	for (float z = selfPos.z - maxDistance; z <= selfPos.z + maxDistance; z += cellWidth)
	{
		for (float y = selfPos.y - maxDistance; y <= selfPos.y + maxDistance; y += cellWidth)
		{
			for (float x = selfPos.x - maxDistance; x <= selfPos.x + maxDistance; x += cellWidth)
			{
				if ( x < gridMin.x || y < gridMin.y || z < gridMin.z)
				{
					continue;
				}

				int posX = (x - gridMin.x) * inverseCellWidth;
				int posY = (y - gridMin.y) * inverseCellWidth;
				int posZ = (z - gridMin.z) * inverseCellWidth;

				// - For each cell, read the start/end indices in the boid pointer array.
				int gridIdx = gridIndex3Dto1D(posX, posY, posZ, gridResolution);
				if (gridIdx < 0 || gridIdx > lastGridIdx)
				{
					continue; // skip if past cell bounds
				}

				int start = gridCellStartIndices[gridIdx];
				int end = gridCellEndIndices[gridIdx];
				if (start == -1 || end == -1)
				{
					continue;
				}

				// - Access each boid in the cell and compute velocity change from
				//   the boids rules, if this boid is within the neighborhood distance.
				for (int neighborIdx = start; neighborIdx <= end; ++neighborIdx)
				{
					if (neighborIdx == index)
					{
						continue;
					}

					glm::vec3 neighborPos = pos[neighborIdx];
					float distance = glm::distance(pos[index], neighborPos);
					// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
					if (distance < rule1Distance)
					{
						cohesionCenter += neighborPos;
						numCohesionNeighbors++;
					}

					// Rule 2: boids try to stay a distance d away from each other, including other boids
					if (distance < rule2Distance)
					{
						separationCenter -= (neighborPos - pos[index]);
					}

					// Rule 3: boids try to match the speed of surrounding boids
					if (distance < rule3Distance)
					{
						alignmentVel += vel1[neighborIdx];
						numAlignmentNeighbors++;
					}
				}
			}
		}
	}

	glm::vec3 new_vel = vel1[index];

	// Rule 1: Cohesion
	if (numCohesionNeighbors > 0)
	{
		cohesionCenter /= numCohesionNeighbors;
		new_vel += (cohesionCenter - pos[index]) * rule1Scale;
	}

	// Rule 2: Separation
	new_vel += separationCenter * rule2Scale;

	// Rule 3: Alignment
	if (numAlignmentNeighbors > 0)
	{
		alignmentVel /= numAlignmentNeighbors;
		new_vel += alignmentVel * rule3Scale;
	}

	// - Clamp the speed change before putting the new speed in vel2
	if (glm::length(new_vel) > maxSpeed)
	{
		new_vel = new_vel / glm::length(new_vel) * maxSpeed;
	}

	vel2[index] = new_vel;
}

/**
 * Step the entire N-body simulation by `dt` seconds.
 */
void Boids::stepSimulationNaive(float dt)
{
	// TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);

	// use dev_vel1
	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);

	// ping-pong the velocity buffers
	hipMemcpy(dev_vel1, dev_vel2, sizeof(int) * numObjects, hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationScatteredGrid(float dt)
{
	// TODO-2.1
	// Uniform Grid Neighbor search using Thrust sort.
	// In Parallel:
	// - label each particle with its array index as well as its grid index.
	//   Use 2x width grids.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	{
		// Or ?
		// hipMemset(dev_gridCellStartIndices, -1, gridCellCount * sizeof(int));
		// hipMemset(dev_gridCellEndIndices, -1, gridCellCount * sizeof(int));

		dim3 perGrid((gridCellCount + blockSize - 1) / blockSize);
		kernResetIntBuffer<<<perGrid, blockSize>>>(gridCellCount,
												   dev_gridCellStartIndices, -1);
		kernResetIntBuffer<<<perGrid, blockSize>>>(gridCellCount,
												   dev_gridCellEndIndices, -1);
	}

	kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

	// - Update positions
	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);

	// - Ping-pong buffers as needed
	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt)
{
	// TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
	// Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
	// In Parallel:
	// - Label each particle with its array index as well as its grid index.
	//   Use 2x width grids
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	{
		dim3 perGrid((gridCellCount + blockSize - 1) / blockSize);
		kernResetIntBuffer<<<perGrid, blockSize>>>(gridCellCount,
												   dev_gridCellStartIndices, -1);
		kernResetIntBuffer<<<perGrid, blockSize>>>(gridCellCount,
												   dev_gridCellEndIndices, -1);
	}
	kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	kernRearrangeBoidData<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleArrayIndices, dev_pos, dev_vel1, dev_sorted_pos, dev_sorted_vel);

	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_sorted_pos, dev_sorted_vel, dev_vel2);

	// - Update positions
	kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_sorted_pos, dev_vel2);

	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
	std::swap(dev_vel1, dev_vel2);
	std::swap(dev_pos, dev_sorted_pos);
}

void Boids::endSimulation()
{
	hipFree(dev_vel1);
	hipFree(dev_vel2);
	hipFree(dev_pos);

	// TODO-2.1 TODO-2.3 - Free any additional buffers here.
	hipFree(dev_particleArrayIndices);
	hipFree(dev_particleGridIndices);
	hipFree(dev_gridCellStartIndices);
	hipFree(dev_gridCellEndIndices);

	hipFree(dev_sorted_pos);
	hipFree(dev_sorted_vel);
}

void Boids::unitTest()
{
	// LOOK-1.2 Feel free to write additional tests here.

	// test unstable sort
	int *dev_intKeys;
	int *dev_intValues;
	int N = 10;

	std::unique_ptr<int[]> intKeys{new int[N]};
	std::unique_ptr<int[]> intValues{new int[N]};

	intKeys[0] = 0;
	intValues[0] = 0;
	intKeys[1] = 1;
	intValues[1] = 1;
	intKeys[2] = 0;
	intValues[2] = 2;
	intKeys[3] = 3;
	intValues[3] = 3;
	intKeys[4] = 0;
	intValues[4] = 4;
	intKeys[5] = 2;
	intValues[5] = 5;
	intKeys[6] = 2;
	intValues[6] = 6;
	intKeys[7] = 0;
	intValues[7] = 7;
	intKeys[8] = 5;
	intValues[8] = 8;
	intKeys[9] = 6;
	intValues[9] = 9;

	hipMalloc((void **)&dev_intKeys, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

	hipMalloc((void **)&dev_intValues, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	std::cout << "before unstable sort: " << std::endl;
	for (int i = 0; i < N; i++)
	{
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// How to copy data to the GPU
	hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
	thrust::device_ptr<int> dev_thrust_values(dev_intValues);
	// LOOK-2.1 Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

	// How to copy data back to the CPU side from the GPU
	hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
	hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");

	std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < N; i++)
	{
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// cleanup
	hipFree(dev_intKeys);
	hipFree(dev_intValues);
	checkCUDAErrorWithLine("hipFree failed!");
	return;
}
